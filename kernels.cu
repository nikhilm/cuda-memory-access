#include "hip/hip_runtime.h"
#include <cstdint>

#include "kernels.h"

__global__ void gpuMemoryReadKernel(uint8_t* __restrict memory, int n) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint8_t accum = 0;
    if (tid < n) {
        accum = memory[tid];
    }
}

void gpuMemoryRead(uint8_t* memory, int n) {
    constexpr int NUM_THREADS = 1024;
    const int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;
    gpuMemoryReadKernel<<<NUM_BLOCKS, NUM_THREADS>>>(memory, n);
}

__global__ void gpuMemoryWriteKernel(uint8_t* __restrict memory, int n) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < n) {
        memory[tid] = 42;
    }
}

void gpuMemoryWrite(uint8_t* memory, int n) {
    constexpr int NUM_THREADS = 1024;
    const int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;
    gpuMemoryWriteKernel<<<NUM_BLOCKS, NUM_THREADS>>>(memory, n);
}
